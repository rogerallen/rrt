#include "hip/hip_runtime.h"
#include "camera.h"
#include "hitable_list.h"
#include "material.h"
#include "ray.h"
#include "scene.h"
#include "sphere.h"
#include "vec3.h"
#include <assert.h>
#include <hiprand/hiprand_kernel.h>
#include <float.h>
#include <iostream>
#include <time.h>

#ifdef COMPILING_FOR_WSL
#define SUPPORTS_CUDA_MEM_PREFETCH_ASYNC 0
#else
#define SUPPORTS_CUDA_MEM_PREFETCH_ASYNC 1
#endif

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const *const func,
                const char *const file, int const line)
{
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result)
                  << " at " << file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

// Matching the C++ code would recurse enough into color() calls that
// it was blowing up the stack, so we have to turn this into a
// limited-depth loop instead.  Later code in the book limits to a max
// depth of 50, so we adapt this a few chapters early on the GPU.
__device__ vec3 color(const ray &r, hitable **world,
                      hiprandState *local_rand_state)
{
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);
    for (int i = 0; i < 50; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered,
                                     local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return vec3(0.0, 0.0, 0.0);
            }
        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f * (unit_direction.y() + 1.0f);
            vec3 c = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return vec3(0.0, 0.0, 0.0); // exceeded recursion
}

__global__ void rand_init(hiprandState *rand_state)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    // Original: Each thread gets same seed, a different sequence number, no
    // offset hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
    // BUGFIX, see Issue#2: Each thread gets different seed, same sequence for
    // performance improvement of about 2x!
    hiprand_init(1984 + pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void
    __launch_bounds__(64, 12) // maxThreadsPerBlock, minBlocksPerMultiprocessor)
    render(vec3 *fb, int max_x, int max_y, int num_samples, camera **cam,
           hitable **world, hiprandState *rand_state)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0, 0, 0);
    for (int s = 0; s < num_samples; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += color(r, world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    col /= float(num_samples);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
}

#define RND (hiprand_uniform(&local_rand_state))

#if 0
__global__ void create_world(hitable **d_hitables, hitable **d_world,
                             camera **d_camera, int image_width,
                             int image_height, hiprandState *rand_state)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        d_hitables[0] = new sphere(vec3(0, -1000.0, -1), 1000,
                               new lambertian(vec3(0.5, 0.5, 0.5)));
        int i = 1;
        for (int a = -11; a < 11; a++) {
            for (int b = -11; b < 11; b++) {
                float choose_mat = RND;
                vec3 center(a + RND, 0.2, b + RND);
                if (choose_mat < 0.8f) {
                    d_hitables[i++] = new sphere(
                        center, 0.2,
                        new lambertian(vec3(RND * RND, RND * RND, RND * RND)));
                }
                else if (choose_mat < 0.95f) {
                    d_hitables[i++] = new sphere(
                        center, 0.2,
                        new metal(vec3(0.5f * (1.0f + RND), 0.5f * (1.0f + RND),
                                       0.5f * (1.0f + RND)),
                                  0.5f * RND));
                }
                else {
                    d_hitables[i++] = new sphere(center, 0.2, new dielectric(1.5));
                }
            }
        }
        d_hitables[i++] = new sphere(vec3(0, 1, 0), 1.0, new dielectric(1.5));
        d_hitables[i++] = new sphere(vec3(-4, 1, 0), 1.0,
                                 new lambertian(vec3(0.4, 0.2, 0.1)));
        d_hitables[i++] =
            new sphere(vec3(4, 1, 0), 1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
        *rand_state = local_rand_state;
        *d_world = new hitable_list(d_hitables, 22 * 22 + 1 + 3);

        for(int j = 0; j < i; j++) {
            d_hitables[j]->print(j);
        }

        vec3 lookfrom(13, 2, 3);
        vec3 lookat(0, 0, 0);
        float dist_to_focus = 10.0;
        (lookfrom - lookat).length();
        float aperture = 0.1;
        *d_camera = new camera(lookfrom, lookat, vec3(0, 1, 0), 30.0,
                               float(image_width) / float(image_height),
                               aperture, dist_to_focus);
    }
}

__global__ void free_world(hitable **d_hitables, hitable **d_world,
                           camera **d_camera)
{
    for (int i = 0; i < 22 * 22 + 1 + 3; i++) {
        delete ((sphere *)d_hitables[i])->mat_ptr;
        delete d_hitables[i];
    }
    delete *d_world;
    delete *d_camera;
}
#else
__global__ void create_world(hitable **d_world, scene_camera *d_scene_camera,
                             camera **d_camera, int num_materials,
                             scene_material *d_scene_materials,
                             material **d_materials, int num_spheres,
                             scene_sphere *d_scene_spheres,
                             hitable **d_hitables, int image_width,
                             int image_height)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {

        vec3 from =
            vec3((*d_scene_camera).lookfrom_x, (*d_scene_camera).lookfrom_y,
                 (*d_scene_camera).lookfrom_z);
        *d_camera = new camera(
            from,
            vec3(d_scene_camera->lookat_x, d_scene_camera->lookat_y,
                 d_scene_camera->lookat_z),
            vec3(d_scene_camera->vup_x, d_scene_camera->vup_y,
                 d_scene_camera->vup_z),
            (float)d_scene_camera->vfov,
            float(image_width) / float(image_height),
            (float)d_scene_camera->aperture, (float)d_scene_camera->focus);

        for (int i = 0; i < num_materials; ++i) {
            scene_material *m = &(d_scene_materials[i]);
            if (m->type == LAMBERTIAN) {
                d_materials[i] = new lambertian(
                    vec3(m->mat.lambertian.albedo_r, m->mat.lambertian.albedo_g,
                         m->mat.lambertian.albedo_b));
            }
            else if (m->type == METAL) {
                d_materials[i] =
                    new metal(vec3(m->mat.metal.albedo_r, m->mat.metal.albedo_g,
                                   m->mat.metal.albedo_b),
                              m->mat.metal.fuzz);
            }
            else if (m->type == DIELECTRIC) {
                d_materials[i] = new dielectric(m->mat.dielectric.ref_idx);
            }
        }

        for (int i = 0; i < num_spheres; ++i) {
            scene_sphere *s = &(d_scene_spheres[i]);
            d_hitables[i] =
                new sphere(vec3(s->center_x, s->center_y, s->center_z),
                           s->radius, d_materials[s->material_index]);
        }

        *d_world = new hitable_list(d_hitables, num_spheres);
    }
}
__global__ void free_world(int num_materials, material **d_materials,
                           int num_spheres, hitable **d_hitables,
                           hitable **d_world, camera **d_camera)
{
    for (int i = 0; i < num_materials; i++) {
        delete d_materials[i];
    }
    for (int i = 0; i < num_spheres; i++) {
        delete d_hitables[i];
    }
    delete *d_world;
    delete *d_camera;
}
#endif

void usage(char *argv)
{
    std::cerr << "Unexpected argument: " << argv << "\n\n";
    std::cerr << "Usage: rrt [options]\n";
    std::cerr << "  -i file.txt         : input scene file\n";
    std::cerr << "  -w <width>          : output image width.  Default: 1200\n";
    std::cerr << "  -h <height>         : output image height.  Default: 800\n";
    std::cerr << "  -s <samples>        : number of samples per pixel.  "
                 "Default: 10\n";
    std::cerr << "  -tx <num_threads_x> : number of threads per block in x.  "
                 "Default: 8\n";
    std::cerr << "  -ty <num_threads_y> : number of threads per block in y.  "
                 "Default: 8\n";
    std::exit(1);
}

int main(int argc, char *argv[])
{

    int image_width = 1200;
    int image_height = 800;
    int num_samples = 10;
    int num_threads_x = 8;
    int num_threads_y = 8;
    scene *the_scene = nullptr;

    for (int i = 1; i < argc; ++i) {
        if (argv[i][0] == '-') {
            if (argv[i][1] == 'w') {
                image_width = atoi(argv[++i]);
            }
            else if (argv[i][1] == 'h') {
                image_height = atoi(argv[++i]);
            }
            else if (argv[i][1] == 's') {
                num_samples = atoi(argv[++i]);
            }
            else if (argv[i][1] == 't') {
                if (argv[i][2] == 'x') {
                    num_threads_x = atoi(argv[++i]);
                }
                else if (argv[i][2] == 'y') {
                    num_threads_y = atoi(argv[++i]);
                }
                else {
                    usage(argv[i]);
                }
            }
            else if (argv[i][1] == 'i') {
                the_scene = new scene(argv[++i]);
            }
            else {
                usage(argv[i]);
            }
        }
        else {
            usage(argv[i]);
        }
    }
    int num_blocks_x = image_width / num_threads_x + 1;
    int num_blocks_y = image_height / num_threads_y + 1;

    int cuda_runtime_version = -1;
    checkCudaErrors(hipRuntimeGetVersion(&cuda_runtime_version));

    std::cerr << "CUDA Runtime Version " << cuda_runtime_version << "\n";
    std::cerr << "Rendering a " << image_width << "x" << image_height
              << " image with " << num_samples << " samples per pixel ";
    std::cerr << "in " << num_blocks_x << "x" << num_blocks_y << " = "
              << num_blocks_x * num_blocks_y << " blocks of " << num_threads_x
              << "x" << num_threads_y << " threads each.\n";

    int num_pixels = image_width * image_height;
    size_t fb_size = num_pixels * sizeof(vec3);

    // allocate FB
    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(
        hipMalloc((void **)&d_rand_state, num_pixels * sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    checkCudaErrors(
        hipMalloc((void **)&d_rand_state2, 1 * sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init<<<1, 1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make our world of hitables & the camera
#if 0
    hitable **d_hitables;
    int num_hitables = 22 * 22 + 1 + 3;
    checkCudaErrors(
        hipMalloc((void **)&d_hitables, num_hitables * sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    create_world<<<1, 1>>>(d_hitables, d_world, d_camera, image_width, image_height,
                           d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
#else
    // create & populate scene data that create_world with use to make the
    // scene.
    scene_camera *d_scene_camera;
    checkCudaErrors(
        hipMallocManaged((void **)&d_scene_camera, sizeof(scene_camera)));
    *d_scene_camera = the_scene->camera;

    scene_material *d_scene_materials;
    int num_materials = the_scene->materials.size();
    checkCudaErrors(hipMallocManaged((void **)&d_scene_materials,
                                      num_materials * sizeof(scene_material)));
    for (int i = 0; i < num_materials; ++i) {
        d_scene_materials[i] = *(the_scene->materials[i]);
    }

    scene_sphere *d_scene_spheres;
    int num_spheres = the_scene->spheres.size();
    checkCudaErrors(hipMallocManaged((void **)&d_scene_spheres,
                                      num_spheres * sizeof(scene_sphere)));
    for (int i = 0; i < num_spheres; ++i) {
        d_scene_spheres[i] = *(the_scene->spheres[i]);
    }

    // now create the data that will contain the world.  create_world populates
    // these
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    material **d_materials;
    checkCudaErrors(
        hipMalloc((void **)&d_materials, num_materials * sizeof(material *)));
    hitable **d_hitables;
    checkCudaErrors(
        hipMalloc((void **)&d_hitables, num_spheres * sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMallocManaged((void **)&d_world, sizeof(hitable *)));

    create_world<<<1, 1>>>(d_world, d_scene_camera, d_camera, num_materials,
                           d_scene_materials, d_materials, num_spheres,
                           d_scene_spheres, d_hitables, image_width,
                           image_height);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
#endif

#if SUPPORTS_CUDA_MEM_PREFETCH_ASYNC == 1
    // Prefetch the FB to the GPU
    int device = -1;
    checkCudaErrors(hipGetDevice(&device));
    std::cerr << "CUDA Device: " << device << std::endl;
    checkCudaErrors(hipMemPrefetchAsync(fb, fb_size, device, NULL));
    checkCudaErrors(hipGetLastError());
#endif

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(num_blocks_x, num_blocks_y);
    dim3 threads(num_threads_x, num_threads_y);
    render_init<<<blocks, threads>>>(image_width, image_height, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(fb, image_width, image_height, num_samples,
                                d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    std::cerr << "stats:" << cuda_runtime_version << "," << image_width << ","
              << image_height << "," << num_samples << "," << num_threads_x
              << "," << num_threads_y << "," << timer_seconds << "\n";

#if SUPPORTS_CUDA_MEM_PREFETCH_ASYNC == 1
    // Prefetch the FB back to the CPU
    checkCudaErrors(hipMemPrefetchAsync(fb, fb_size, hipCpuDeviceId, NULL));
    checkCudaErrors(hipGetLastError());
#endif

    // Output FB as Image
    std::cout << "P3\n" << image_width << " " << image_height << "\n255\n";
    for (int j = image_height - 1; j >= 0; j--) {
        for (int i = 0; i < image_width; i++) {
            size_t pixel_index = j * image_width + i;
            int ir = int(255.99 * fb[pixel_index].r());
            int ig = int(255.99 * fb[pixel_index].g());
            int ib = int(255.99 * fb[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
#if 0
    free_world<<<1, 1>>>(d_hitables, d_world, d_camera);
#else
    free_world<<<1, 1>>>(num_materials, d_materials, num_spheres, d_hitables,
                         d_world, d_camera);
#endif
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_scene_camera));
    checkCudaErrors(hipFree(d_scene_materials));
    checkCudaErrors(hipFree(d_scene_spheres));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(fb));

    if (the_scene) {
        delete the_scene;
    }

    hipDeviceReset();
}
